#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
/*************************************************************************************************
 * Copyright (C) 2017, Nikhil Hegde, Jianqiao Liu, Kirshanthan Sundararajah, Milind Kulkarni, and 
 * Purdue University. All Rights Reserved. See Copyright.txt
*************************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "util_common.h"

#include "bh_kernel.h"

__global__ void init_kernel(void) {

}

__global__ void compute_force_gpu(bh_gpu_tree root, Point* points, int npoints, float eps_squared, float dthf, int step) {
	
	unsigned int i, pidx, j;
	int cur_node_index;
  
	Vec a_prev;
	Vec p_cofm;
	Vec p_acc;
	Vec p_vel;
	Vec delta_v;
	Vec dr;
	float drsq;
	float idr;
	float nphi;
	float scale;
	float ropensq;
	
	gpu_node0 cur_node0;
	gpu_node1 cur_node1;
	gpu_node2 cur_node2;

	#ifdef TRACK_TRAVERSALS
	int nodes_accessed;
	#endif
	
	int stk[128];
	int stk_top;
	int sp; 

	// loop over all points for this node
	for(pidx = blockIdx.x*blockDim.x + threadIdx.x; pidx < npoints; pidx += gridDim.x*blockDim.x) {
				
		// Cache this threads point
		//p_mass = points[pidx].mass;
		p_cofm = points[pidx].cofm;
		p_acc = points[pidx].acc;
		#ifdef TRACK_TRAVERSALS
		nodes_accessed = 0;
		#endif
		a_prev = p_acc;
		p_acc.x = 0.0;
		p_acc.y = 0.0;
		p_acc.z = 0.0;

		STACK_INIT();

		while(sp >= 1) {
			cur_node_index = STACK_TOP_NODE_INDEX;				
			STACK_POP();

			#ifdef TRACK_TRAVERSALS
			nodes_accessed++;
			#endif
			//CUR_NODE0 = params.root.nodes0[cur_node_index];
			dr.x = root.nodes0[cur_node_index].cofm.x - p_cofm.x;
			dr.y = root.nodes0[cur_node_index].cofm.y - p_cofm.y;
			dr.z = root.nodes0[cur_node_index].cofm.z - p_cofm.z;
			drsq = dr.x*dr.x + dr.y*dr.y + dr.z*dr.z;
			ropensq = root.nodes1[cur_node_index].ropen * root.nodes1[cur_node_index].ropen;
			 
			//params.root.nodes1[cur_node_index] = params.root.nodes1[cur_node_index];
			if(drsq < ropensq) {
				if (root.nodes0[cur_node_index].leafNode == false) {			
					if (root.nodes2[cur_node_index].right != -1) {
						STACK_PUSH();
						STACK_TOP_NODE_INDEX = root.nodes2[cur_node_index].right;
					}
					if (root.nodes2[cur_node_index].left != -1) {
						STACK_PUSH();
						STACK_TOP_NODE_INDEX = root.nodes2[cur_node_index].left;
					}
				} else {
					if(points[pidx].id != root.nodes0[cur_node_index].point_id) {
						//drsq_2 = drsq + epssq;
						drsq += 2*eps_squared;
						idr = rsqrtf(drsq); //1.0 / sqrt(drsq);
						nphi = root.nodes1[cur_node_index].mass * idr;
						scale = nphi * idr *idr;

						p_acc.x += dr.x*scale;
						p_acc.y += dr.y*scale;
						p_acc.z += dr.z*scale;
					}
				}
			} else {
				//drsq_2 = drsq + epssq;
				drsq += 2*eps_squared;
				idr = rsqrtf(drsq); //1.0 / sqrt(drsq);
				nphi = root.nodes1[cur_node_index].mass * idr;
				scale = nphi * idr * idr;
							
				p_acc.x += dr.x*scale;
				p_acc.y += dr.y*scale;
				p_acc.z += dr.z*scale;
			}
		}

	  	p_vel = points[pidx].vel;
		if(step > 0) {
			
		  delta_v.x = (p_acc.x - a_prev.x) * dthf;
		  delta_v.y = (p_acc.y - a_prev.y) * dthf;
		  delta_v.z = (p_acc.z - a_prev.z) * dthf;
		  
		  p_vel.x = p_vel.x + delta_v.x;
		  p_vel.y = p_vel.y + delta_v.y;
		  p_vel.z = p_vel.z + delta_v.z;
		}
	       
		// Write cached point back to tree
		points[pidx].vel = p_vel;
		points[pidx].acc = p_acc;
		#ifdef TRACK_TRAVERSALS
		points[pidx].num_nodes_traversed = nodes_accessed;
		#endif
	}
	
}


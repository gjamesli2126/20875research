#include "hip/hip_runtime.h"
/*************************************************************************************************
 * Copyright (C) 2017, Nikhil Hegde, Jianqiao Liu, Kirshanthan Sundararajah, Milind Kulkarni, and
 * Purdue University. All Rights Reserved. See Copyright.txt
*************************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>
#include <time.h>
#include "timer.c"

#include "common.h"
#include "kd.cu"

#if defined (GPU) || defined (GPU_KD)
// From the NVIDIA CUDA programming guide. No idea how it works
__device__ double atomicAdd(double* address, double val) 
{ 
    double old = *address, assumed; 
    do {
        assumed = old;
        old = __longlong_as_double(atomicCAS((unsigned long long int*)address,
                                             __double_as_longlong(assumed),
                                             __double_as_longlong(val + assumed)));
    } while (assumed != old); 
    return old;
}

__global__ void ResetCentroidForEachCluster(Cluster *clusters)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;
    // Set clusters[].noOfPoints to 0. Only for K centroids
    if (pt < K) {
        clusters[pt].noOfPoints = 0;
    }
}

__global__ void ComputeClusters(Point *points, Cluster *clusters, Point *tempPoints)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;
    int i; double max; int inCluster;

    if (pt >= N)
        return;

    // Save the old centroid and clear the x and y components of
    // each point. We're going to use first K of these to store 
    // the sum of co-ordinates of points in this cluster.
    // clusterId field is used to save old centroid for each point
    // so that we know when to stop iterating.
    tempPoints[pt].clusterId = points[pt].clusterId;
    tempPoints[pt].loc[X_AXIS] = 0.0;
    tempPoints[pt].loc[Y_AXIS] = 0.0;

    // Compute the nearest centroid.
    max = GetDistanceGPU(points[pt], clusters[0].pt);
    inCluster = 0;
    for (i = 0; i < K; i++) {
        if (GetDistanceGPU(points[pt], clusters[i].pt) < max) {
            inCluster = i;
            max = GetDistanceGPU(points[pt], clusters[i].pt);
        }
    }
    atomicAdd(&clusters[inCluster].noOfPoints, 1);
    // Bottle neck I'm sure.
    atomicAdd(&tempPoints[inCluster].loc[X_AXIS], points[pt].loc[X_AXIS]);
    atomicAdd(&tempPoints[inCluster].loc[Y_AXIS], points[pt].loc[Y_AXIS]);
    
    points[pt].clusterId = inCluster;
}

__global__ void ComputeCentroids(Cluster *clusters, Point *tempPoints)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;

    // Now calculate the new centroids.
    if (pt < K) {
        clusters[pt].pt.loc[X_AXIS] = tempPoints[pt].loc[X_AXIS]/clusters[pt].noOfPoints;
        clusters[pt].pt.loc[Y_AXIS] = tempPoints[pt].loc[Y_AXIS]/clusters[pt].noOfPoints;
    }
}

__global__ void RepeatNeeded(Point *points, Point *tempPoints, unsigned int *key)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;

    if (pt < N) {
        if (points[pt].clusterId != tempPoints[pt].clusterId) {
                *key = 1;
        }
    }

}

#if defined (GPU_KD)
__global__ void ComputeClustersKdTree(Point *points, Cluster *clusters, Point *tempPoints, KdTree *kdTree, char *visitedPerThread)
{
    int pt = blockIdx.x*blockDim.x + threadIdx.x;
    int inCluster;
    Cluster *nearestCluster;
    char *visitedForThisThread = visitedPerThread +(K*pt);

    if (pt >= N)
        return;

    // Save the old centroid and clear the x and y components of
    // each point. We're going to use first K of these to store 
    // the sum of co-ordinates of points in this cluster.
    // clusterId field is used to save old centroid for each point
    // so that we know when to stop iterating.
    tempPoints[pt].clusterId = points[pt].clusterId;
    tempPoints[pt].loc[X_AXIS] = 0.0;
    tempPoints[pt].loc[Y_AXIS] = 0.0;

    // Compute the nearest centroid.
    nearestCluster = NearestNeighbourGPU
        (kdTree, clusters, visitedForThisThread, points[pt], K);
    inCluster = nearestCluster->pt.clusterId;

    atomicAdd(&clusters[inCluster].noOfPoints, 1);
    // Bottle neck I'm sure.
    atomicAdd(&tempPoints[inCluster].loc[X_AXIS], points[pt].loc[X_AXIS]);
    atomicAdd(&tempPoints[inCluster].loc[Y_AXIS], points[pt].loc[Y_AXIS]);
    
    points[pt].clusterId = inCluster;
}

void DoKmeansGPUKdTree (Point *points, Cluster *clusters)
{

    Point *dPoints, *dTempPoints; 
    Cluster *dClusters; unsigned int *repeat, repeatHost;
    KdTree *kdTree, *dKdTree;
    char *visitedPerThread;

    hipMalloc ((void **)&dPoints, sizeof(Point)*N);
    hipMalloc ((void **)&dClusters, sizeof(Cluster)*K);
    hipMalloc ((void **)&dTempPoints, sizeof(Point)*N);
    hipMalloc ((void **)&repeat, sizeof(unsigned int));
    hipMalloc ((void **)&visitedPerThread, sizeof(char)*K*N);
    hipMalloc ((void **)&dKdTree, sizeof(KdTree)*K);

    hipMemcpy(dPoints, points, sizeof(Point)*N, hipMemcpyHostToDevice);
    hipMemcpy(dClusters, clusters, sizeof(Cluster)*K, hipMemcpyHostToDevice);

    dim3 threadsPerBlock (256);
    dim3 blocksPerGrid (N/threadsPerBlock.x);

    do {
        ResetCentroidForEachCluster<<<blocksPerGrid, threadsPerBlock>>>(dClusters);

        // Copy kdtree between host and device. (KdTree computation is done in host).
        hipMemcpy(clusters, dClusters, sizeof(Cluster)*K, hipMemcpyDeviceToHost);
        kdTree = BuildKdTree(clusters, K, false);
        hipMemcpy(dKdTree, kdTree, sizeof(KdTree)*K, hipMemcpyHostToDevice);
        ComputeClustersKdTree<<<blocksPerGrid, threadsPerBlock>>>
            (dPoints, dClusters, dTempPoints, dKdTree, visitedPerThread);

        ComputeCentroids<<<blocksPerGrid, threadsPerBlock>>>(dClusters, dTempPoints);
        hipMemset(repeat, 0, sizeof(unsigned int));
        RepeatNeeded<<<blocksPerGrid, threadsPerBlock>>>(dPoints, dTempPoints, repeat);
        hipMemcpy(&repeatHost, repeat, sizeof(unsigned int), hipMemcpyDeviceToHost);
    } while (repeatHost);
    
    
    hipMemcpy(points, dPoints, sizeof(Point)*N, hipMemcpyDeviceToHost);
    hipMemcpy(clusters, dClusters, sizeof(Cluster)*K, hipMemcpyDeviceToHost);

    hipFree(dPoints);
    hipFree(dClusters);
    hipFree(dTempPoints);
    hipFree(repeat);
    hipFree(visitedPerThread);
    hipFree(dKdTree);
}
#endif // defined (GPU_KD)

void DoKmeansGPU (Point *points, Cluster *clusters)
{

    Point *dPoints, *dTempPoints; 
    Cluster *dClusters; unsigned int *repeat, repeatHost;

    hipMalloc ((void **)&dPoints, sizeof(Point)*N);
    hipMalloc ((void **)&dClusters, sizeof(Cluster)*K);
    hipMalloc ((void **)&dTempPoints, sizeof(Point)*N);
    hipMalloc ((void **)&repeat, sizeof(unsigned int));

    hipMemcpy(dPoints, points, sizeof(Point)*N, hipMemcpyHostToDevice);
    hipMemcpy(dClusters, clusters, sizeof(Cluster)*K, hipMemcpyHostToDevice);

    dim3 threadsPerBlock (256);
    dim3 blocksPerGrid (N/threadsPerBlock.x);

    do {
        ResetCentroidForEachCluster<<<blocksPerGrid, threadsPerBlock>>>(dClusters);
        ComputeClusters<<<blocksPerGrid, threadsPerBlock>>>(dPoints, dClusters, dTempPoints);
        ComputeCentroids<<<blocksPerGrid, threadsPerBlock>>>(dClusters, dTempPoints);
        hipMemset(repeat, 0, sizeof(unsigned int));
        RepeatNeeded<<<blocksPerGrid, threadsPerBlock>>>(dPoints, dTempPoints, repeat);
        hipMemcpy(&repeatHost, repeat, sizeof(unsigned int), hipMemcpyDeviceToHost);
    } while (repeatHost);
    
    
    hipMemcpy(points, dPoints, sizeof(Point)*N, hipMemcpyDeviceToHost);
    hipMemcpy(clusters, dClusters, sizeof(Cluster)*K, hipMemcpyDeviceToHost);

    hipFree(dPoints);
    hipFree(dClusters);
    hipFree(dTempPoints);
    hipFree(repeat);
}
#endif // defined(GPU) || defined(GPU_KD)

#if defined (CPU_KD)
void DoKmeansCPUKdTree (Point *points, Cluster *clusters)
{
    int i, inCluster;
    bool changed;
    Point *tempPoints;
    Cluster *nearestCluster;
    KdTree *kdTree;

    // One for each cluster (and point). For cluster, use x and y, for point, clusterId.
    tempPoints  = (Point *) malloc (sizeof(Point)*N);
   
    do {

        memset(tempPoints, 0, sizeof(Point)*N);

        for (i = 0; i < K; i++) {
            clusters[i].noOfPoints = 0;
        }
        // Save the old clusterId for each point. Reusing tempPoints
        for (i = 0; i < N; i++) {
            tempPoints[i].clusterId = points[i].clusterId;
        }
        
        kdTree = BuildKdTree(clusters, K, false);
        // For each point, find the nearest centroid.
        for (i = 0; i < N; i++) {
            nearestCluster = NearestNeighbour(kdTree, clusters, points[i]);
            inCluster = nearestCluster->pt.clusterId;
            clusters[inCluster].noOfPoints++;
            tempPoints[inCluster].loc[X_AXIS] += points[i].loc[X_AXIS];
            tempPoints[inCluster].loc[Y_AXIS] += points[i].loc[Y_AXIS];
            points[i].clusterId = inCluster;
        }

        // Compute new centroid for each cluster
        for (i = 0; i < K; i++) {
            // Assuming that each cluster has atleast one point in it.
            assert(clusters[i].noOfPoints != 0);
            clusters[i].pt.loc[X_AXIS] = tempPoints[i].loc[X_AXIS]/clusters[i].noOfPoints;
            clusters[i].pt.loc[Y_AXIS] = tempPoints[i].loc[Y_AXIS]/clusters[i].noOfPoints;
        }

        // Check if anything has changed
        changed = false;
        for (i = 0; i < N; i++) {
            if (points[i].clusterId != tempPoints[i].clusterId) {
                changed = true;
                break;
            }
        }
    } while (changed);

}
#endif // #if defined (CPU_KD)

void DoKmeansCPU (Point *points, Cluster *clusters)
{

    double max;
    int i, j, inCluster;
    bool changed;
    Point *tempPoints;

    // One for each cluster (and point). For cluster, use x and y, for point, clusterId.
    tempPoints  = (Point *) malloc (sizeof(Point)*N);
   
    do {

        memset(tempPoints, 0, sizeof(Point)*N);

        for (i = 0; i < K; i++) {
            clusters[i].noOfPoints = 0;
        }
        // Save the old clusterId for each point. Reusing tempPoints
        for (i = 0; i < N; i++) {
            tempPoints[i].clusterId = points[i].clusterId;
        }
        // For each point, find the nearest centroid.
        for (i = 0; i < N; i++) {
            max = GetDistance(points[i], clusters[0].pt);
            inCluster = 0;
            for (j = 0; j < K; j++) {
                if (GetDistance(points[i], clusters[j].pt) < max) {
                    inCluster = j;
                    // TODO: We should next store these distances, instead of re-computing
                    // (I don't mean from above call, I mean totally for the program).
                    max = GetDistance(points[i], clusters[j].pt);
                }
            }
            clusters[inCluster].noOfPoints++;
            tempPoints[inCluster].loc[X_AXIS] += points[i].loc[X_AXIS];
            tempPoints[inCluster].loc[Y_AXIS] += points[i].loc[Y_AXIS];
            points[i].clusterId = inCluster;
        }

        // Compute new centroid for each cluster
        for (i = 0; i < K; i++) {
            // Assuming that each cluster has atleast one point in it.
            assert(clusters[i].noOfPoints != 0);
            clusters[i].pt.loc[X_AXIS] = tempPoints[i].loc[X_AXIS]/clusters[i].noOfPoints;
            clusters[i].pt.loc[Y_AXIS] = tempPoints[i].loc[Y_AXIS]/clusters[i].noOfPoints;
        }

        // Check if anything has changed
        changed = false;
        for (i = 0; i < N; i++) {
            if (points[i].clusterId != tempPoints[i].clusterId) {
                changed = true;
                break;
            }
        }
    } while (changed);

}

int main (int argc, char *argv[])
{

    Point *pointsCPU;
    Cluster *clustersCPU;
    int i, j;

#ifdef GPU
    Point *pointsGPU;
    Cluster *clustersGPU;
#endif

#ifdef GPU_KD
    Point *pointsGPUKdTree;
    Cluster *clustersGPUKdTree;
#endif

#ifdef CPU_KD
    Point *pointsCPUKdTree;
    Cluster *clustersCPUKdTree;
#endif

    srandom(time(NULL));

    pointsCPU = (Point *) malloc (sizeof(Point)*N);
    clustersCPU = (Cluster *) malloc (sizeof(Cluster)*K);

    // Get the points randomly
    for (i = 0; i < N; i++) {
        pointsCPU[i].loc[X_AXIS] = (random()/1021322);
        pointsCPU[i].loc[Y_AXIS] = (random()/1021322);
        pointsCPU[i].clusterId = -1;
    }

    // Initialize clusters
    for (i = 0; i < K; i++) {
        clustersCPU[i].pt.clusterId = i;
        clustersCPU[i].noOfPoints = 0;
        j = random()%N;
        if (pointsCPU[j].clusterId != -1) {
            i--; continue;
            // Potential infinite loop
        }
        pointsCPU[j].clusterId = i;
        clustersCPU[i].pt.loc[X_AXIS] = pointsCPU[j].loc[X_AXIS];
        clustersCPU[i].pt.loc[Y_AXIS] = pointsCPU[j].loc[Y_AXIS];
    }

#ifdef DEBUG
    printf ("Initial points:\n");
    for (i = 0; i < N; i++) {
        printf ("x=%.2f,y=%.2f,clusterId=%d\n", pointsCPU[i].loc[X_AXIS], pointsCPU[i].loc[Y_AXIS], pointsCPU[i].clusterId);
    }
    printf ("Initial clusters:\n");
    for (i = 0; i < K; i++) {
        printf("clusterId=%d,noOfPoints=%d,centroidX=%.2f,centroidY=%.2f\n", clustersCPU[i].pt.clusterId, 
               clustersCPU[i].noOfPoints, clustersCPU[i].pt.loc[X_AXIS], clustersCPU[i].pt.loc[Y_AXIS]);
    }
#endif // DEBUG

#ifdef GPU_KD
    pointsGPUKdTree = (Point *) malloc (sizeof(Point)*N);
    clustersGPUKdTree = (Cluster *) malloc (sizeof(Cluster)*K);

    memcpy(pointsGPUKdTree, pointsCPU, sizeof(Point)*N);
    memcpy(clustersGPUKdTree, clustersCPU, sizeof(Cluster)*K);

    tstart();
    DoKmeansGPUKdTree(pointsGPUKdTree, clustersGPUKdTree);
    tend();
    printf("%f seconds on GPU KdTree.\n", tval());
#endif 

#ifdef GPU
    pointsGPU = (Point *) malloc (sizeof(Point)*N);
    clustersGPU = (Cluster *) malloc (sizeof(Cluster)*K);

    memcpy(pointsGPU, pointsCPU, sizeof(Point)*N);
    memcpy(clustersGPU, clustersCPU, sizeof(Cluster)*K);

    tstart();
    DoKmeansGPU(pointsGPU, clustersGPU);
    tend();
    printf("%f seconds on GPU.\n", tval());
#endif 

#ifdef CPU_KD
    pointsCPUKdTree = (Point *) malloc (sizeof(Point)*N);
    clustersCPUKdTree = (Cluster *) malloc (sizeof(Cluster)*K);

    memcpy(pointsCPUKdTree, pointsCPU, sizeof(Point)*N);
    memcpy(clustersCPUKdTree, clustersCPU, sizeof(Cluster)*K);

    tstart();
    DoKmeansCPUKdTree(pointsCPUKdTree, clustersCPUKdTree);
    tend();
    printf("%f seconds on CPU KdTree.\n", tval());
#endif

    // Note plain CPU should always be at the end. Data for other versions are
    // copied from here. So don't want it to change before copying.
    tstart();
    DoKmeansCPU(pointsCPU, clustersCPU);
    tend();
    printf("%f seconds on CPU.\n", tval());


#ifdef PRETTY_PRINT
#if defined (GPU_KD)
    // Showing GPU_KD dumps
    FILE *fp; char buf[20];
    system ("rm /tmp/*plot");
    for (i = 0; i < N; i++) {
        sprintf(buf, "/tmp/%d.plot", pointsGPUKdTree[i].clusterId);
        fp = fopen (buf, "a");
        if (fp) {
            fprintf (fp, "%.2f %.2f #%d GPUKd\n", pointsGPUKdTree[i].loc[X_AXIS], pointsGPUKdTree[i].loc[Y_AXIS], pointsGPUKdTree[i].clusterId);
            fclose(fp);
        }
    }
#elif defined (GPU)
    // Showing GPU dumps
    FILE *fp; char buf[20];
    system ("rm /tmp/*plot");
    for (i = 0; i < N; i++) {
        sprintf(buf, "/tmp/%d.plot", pointsGPU[i].clusterId);
        fp = fopen (buf, "a");
        if (fp) {
            fprintf (fp, "%.2f %.2f #%d GPU\n", pointsGPU[i].loc[X_AXIS], pointsGPU[i].loc[Y_AXIS], pointsGPU[i].clusterId);
            fclose(fp);
        }
    }
#elif defined(CPU_KD)
    // Showing CPU_KD dumps
    FILE *fp; char buf[20];
    system ("rm /tmp/*plot");
    for (i = 0; i < N; i++) {
        sprintf(buf, "/tmp/%d.plot", pointsCPUKdTree[i].clusterId);
        fp = fopen (buf, "a");
        if (fp) {
            fprintf (fp, "%.2f %.2f #%d CPUKd\n", pointsCPUKdTree[i].loc[X_AXIS], pointsCPUKdTree[i].loc[Y_AXIS], pointsCPUKdTree[i].clusterId);
            fclose(fp);
        }
    }
#else
    // Showing CPU dumps
    FILE *fp; char buf[20];
    system ("rm /tmp/*plot");
    for (i = 0; i < N; i++) {
        sprintf(buf, "/tmp/%d.plot", pointsCPU[i].clusterId);
        fp = fopen (buf, "a");
        if (fp) {
            fprintf (fp, "%.2f %.2f #%d CPU\n", pointsCPU[i].loc[X_AXIS], pointsCPU[i].loc[Y_AXIS], pointsCPU[i].clusterId);
            fclose(fp);
        }
    }
#endif // #if defined (GPU_KD)
#endif // PRETTY_PRINT

    return 0;
}

#if 0
/********** Pretty print script ***********
// string=""
// for plot in /tmp/*.plot
// do
//     string="${string},\"$plot\""
// done
//
// string=`cut -c 2- <<EOF
/  $string
// EOF`
// 
// echo "set key off" > /tmp/plot
// echo "plot $string" >> /tmp/plot
// gnuplot -persist < /tmp/plot
// 
// # ah
************** End script ****************/
#endif

/*************************************************************************************************
 * Copyright (C) 2017, Nikhil Hegde, Jianqiao Liu, Kirshanthan Sundararajah, Milind Kulkarni, and
 * Purdue University. All Rights Reserved. See Copyright.txt
*************************************************************************************************/
#ifndef KD_C
#define KD_C

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <stdbool.h>
#include <string.h>

#include "kd.h"
#include "common.h"

#if defined(CPU_KD) || defined (GPU_KD)

static KdTree *KdTreeMemPool = NULL;
// Below array is sorted, rather than actual points.
static int *clusterIdArr = NULL;
struct SortWorkList {
    int from;
    int to;
} *sortWorkList = NULL;

// Number of elements in previous kd tree built.
static int memAllocated = 0;
static int nextFreeNode;

static void QuickSort(const Cluster *nodes, int *idArr, int fFrom, int fTo, int axis) {
    
    int from, to, workListTop, nextEmptySlot, i, temp;

    assert(sortWorkList);
    workListTop = 0;

    sortWorkList[workListTop].from = fFrom;
    sortWorkList[workListTop].to = fTo;

    while (workListTop >= 0) {
        from = sortWorkList[workListTop].from;
        to = sortWorkList[workListTop].to;
        workListTop--;
        // TO is always the pivot.

        assert(from < to);
        nextEmptySlot = from;
        for (i = from; i < to; i++) {
            if (nodes[idArr[i]].pt.loc[axis] <= nodes[idArr[to]].pt.loc[axis]) {
                temp = idArr[nextEmptySlot];
                idArr[nextEmptySlot] = idArr[i];
                idArr[i] = temp;
                nextEmptySlot++;
            }
        }
        temp = idArr[to];
        idArr[to] = idArr[nextEmptySlot];
        idArr[nextEmptySlot] = temp;

        // idArr[nextEmptySlot] now contains "to", the pivot.
        if (from < nextEmptySlot-1) {
            workListTop++;
            sortWorkList[workListTop].from = from;
            sortWorkList[workListTop].to = nextEmptySlot-1;
        }
        if (nextEmptySlot+1 < to) {
            workListTop++;
            sortWorkList[workListTop].from = nextEmptySlot+1;
            sortWorkList[workListTop].to = to;
        }
    }
   
#ifdef DEBUG
    for (i = fFrom; i <= fTo; i++) {
        if (i < fTo) {
            assert(nodes[idArr[i]].pt.loc[axis] <= nodes[idArr[i+1]].pt.loc[axis]);
        }
    }
#endif
}

static int BuildKdTreeRec(Cluster *nodes, int *idArr, int from, int to, int depth) {
    
    KdTree *currentNode, *childNode;
    int currentNodeIndex, median;

    if (from == to) {
        currentNodeIndex = nextFreeNode++;
        currentNode = &KdTreeMemPool[currentNodeIndex];
        currentNode->splitPlane = depth%DIMENSIONS;
        currentNode->clusterId = idArr[from];
        currentNode->leftChildIndex = -1;
        currentNode->rightChildIndex = -1;
        return currentNodeIndex;
    }

    QuickSort(nodes, idArr, from, to, depth%DIMENSIONS);

    currentNodeIndex = nextFreeNode++;
    currentNode = &KdTreeMemPool[currentNodeIndex];
    currentNode->splitPlane = depth%DIMENSIONS;
    // Current node will be the median
    median = (from+to)/2;
    currentNode->clusterId = idArr[median];

    if (median != from) {
        currentNode->leftChildIndex = BuildKdTreeRec(nodes, idArr, from, median-1, depth+1);
    } else {
        currentNode->leftChildIndex = -1;
    }
    currentNode->rightChildIndex = BuildKdTreeRec(nodes, idArr, median+1, to, depth+1);

    if (currentNode->leftChildIndex != -1) {
        childNode = &KdTreeMemPool[currentNode->leftChildIndex];
        childNode->parentIndex = currentNodeIndex;
    }
    assert(currentNode->rightChildIndex != -1);
    childNode = &KdTreeMemPool[currentNode->rightChildIndex];
    childNode->parentIndex = currentNodeIndex;


    return currentNodeIndex;
}

#ifdef DEBUG
void DumpKdTree(Cluster *nodes, KdTree *tree) {

    printf("%d [label=%c]\n", tree->clusterId, 
           (tree->splitPlane == X_AXIS) ? 'X' : 'Y');
    if(tree->leftChildIndex != -1) {
        printf("%d -> %d\n", tree->clusterId,
               KdTreeMemPool[tree->leftChildIndex].clusterId);
        DumpKdTree(nodes, &KdTreeMemPool[tree->leftChildIndex]);
    }
    if (tree->rightChildIndex != -1) {
        printf("%d -> %d\n", tree->clusterId, 
               KdTreeMemPool[tree->rightChildIndex].clusterId);
        DumpKdTree(nodes, &KdTreeMemPool[tree->rightChildIndex]);
    }

}
#endif

KdTree *BuildKdTree(Cluster *nodes, int noOfNodes, bool ReAlloc) {

    KdTree *root;
    int i, rootIndex;

    if (memAllocated < noOfNodes || ReAlloc) {
        if (memAllocated > 0) {
            if (!ReAlloc) {
                free(KdTreeMemPool);
            }
            free(clusterIdArr);
            free(sortWorkList);
        }
        KdTreeMemPool = (KdTree *) malloc (sizeof(KdTree)*noOfNodes);
        clusterIdArr = (int *) malloc (sizeof(int)*noOfNodes);
        sortWorkList = (struct SortWorkList *) malloc (sizeof(struct SortWorkList)*noOfNodes);
        memAllocated = noOfNodes;
    }
    
    for (i = 0; i < noOfNodes; i++) {
        clusterIdArr[i] = i;
        assert(clusterIdArr[i] == nodes[i].pt.clusterId);
    }

    nextFreeNode = 0;
    //printf("noOfNodes = %d\n", noOfNodes);
    rootIndex = BuildKdTreeRec(nodes, clusterIdArr, 0, noOfNodes-1, 0);
    root = &KdTreeMemPool[rootIndex];
    root->parentIndex = -1;

#ifdef DEBUG
    printf("digraph KdTree {\n");
    DumpKdTree(nodes, root);
    printf("}\n");
#endif

    return root;
}
#endif // defined(CPU_KD) || defined (GPU_KD)

#ifdef GPU_KD
__device__ Cluster *NearestNeighbourGPU(KdTree *kdTree, Cluster *nodes, char *visited, Point point, int noOfNodes) {

    int cursor, prevCursor, axis, bestNode;
    double bestDist, subRoot, dist;
    KdTree *KdTreeMemPool = kdTree;

    // assert(visited);
    memset(visited, 0, sizeof(char)*noOfNodes);
    bestDist = -1.0;
    bestNode = -1;

    subRoot = 0;

    // cursor, prevCursor can be cosidered pointers to KdTree nodes.
    // KdTreeMemPool[cursor] is an indirection to that node.
    // KdTreeMemPool[cursor].clusterId is a pointer to the cluster (centroid) that the 
    // node represents. nodes[KdTreeMemPool[cursor].clusterId] is an indrection into the
    // actual cluster node.

    while (1) {

        prevCursor = -1;
        cursor = subRoot;
        
        // Traverse down the binary (kd) tree to the nearest leaf.
        while (cursor != -1) {
            axis = KdTreeMemPool[cursor].splitPlane;
            prevCursor = cursor;
            // TODO: Perf- possibly eliminate below divergence.
            // if point(x/y) <= cursor(x/y), take left path.
            if (point.loc[axis] <= nodes[KdTreeMemPool[cursor].clusterId].pt.loc[axis]) {
                cursor = KdTreeMemPool[cursor].leftChildIndex;
            } else {
                // else right path.
                cursor = KdTreeMemPool[cursor].rightChildIndex;
            }
        }

        // Now unwind the stack
        cursor = prevCursor; // cursor now points to the leaf node reached. 
        prevCursor = -1;
        while (cursor != -1) {
            if (!visited[cursor]) {
                visited[cursor] = 1;
                dist = GetDistanceGPU(point, nodes[KdTreeMemPool[cursor].clusterId].pt);
                if (dist < bestDist || bestNode == -1) {
                    bestNode = cursor;
                    bestDist = dist;
                }

                // See if sibling subtree needs to be visited
                axis = KdTreeMemPool[cursor].splitPlane;
                dist = MOD(nodes[KdTreeMemPool[cursor].clusterId].pt.loc[axis]-point.loc[axis]);
                if (dist < bestDist) {
                    // Traverse down the the other subtree.
                    if (KdTreeMemPool[cursor].leftChildIndex == prevCursor && 
                        KdTreeMemPool[cursor].rightChildIndex != -1)
                    {
                        subRoot = KdTreeMemPool[cursor].rightChildIndex;
                        break;
                    } else if (KdTreeMemPool[cursor].leftChildIndex != -1) {
                        subRoot = KdTreeMemPool[cursor].leftChildIndex;
                        break;
                    }
                }
            }
            // cursor = parent(cursor)
            prevCursor = cursor;
            cursor = KdTreeMemPool[cursor].parentIndex;
        }
        if (cursor == -1)
            break;
        // Else, traverse down new subRoot and up again.
    }
    // assert(bestNode != -1 && bestDist >= 0);
    return &nodes[KdTreeMemPool[bestNode].clusterId];
}
#endif // GPU_KD

#ifdef CPU_KD
Cluster *NearestNeighbour(KdTree *kdTree, Cluster *nodes, Point point) {

    int cursor, prevCursor, axis, bestNode;
    double bestDist, subRoot, dist;
    KdTree *KdTreeMemPool = kdTree;
    int *visited = clusterIdArr;

    assert(visited);
    memset(visited, 0, sizeof(int)*memAllocated);
    bestDist = -1.0;
    bestNode = -1;

    subRoot = 0;

    // cursor, prevCursor can be cosidered pointers to KdTree nodes.
    // KdTreeMemPool[cursor] is an indirection to that node.
    // KdTreeMemPool[cursor].clusterId is a pointer to the cluster (centroid) that the 
    // node represents. nodes[KdTreeMemPool[cursor].clusterId] is an indrection into the
    // actual cluster node.

    while (1) {

        prevCursor = -1;
        cursor = subRoot;
        
        // Traverse down the binary (kd) tree to the nearest leaf.
        while (cursor != -1) {
            axis = KdTreeMemPool[cursor].splitPlane;
            prevCursor = cursor;
            // TODO: Perf- possibly eliminate below divergence.
            // if point(x/y) <= cursor(x/y), take left path.
            if (point.loc[axis] <= nodes[KdTreeMemPool[cursor].clusterId].pt.loc[axis]) {
                cursor = KdTreeMemPool[cursor].leftChildIndex;
            } else {
                // else right path.
                cursor = KdTreeMemPool[cursor].rightChildIndex;
            }
        }

        // Now unwind the stack
        cursor = prevCursor; // cursor now points to the leaf node reached. 
        prevCursor = -1;
        while (cursor != -1) {
            if (!visited[cursor]) {
                visited[cursor] = 1;
                dist = GetDistance(point, nodes[KdTreeMemPool[cursor].clusterId].pt);
                if (dist < bestDist || bestNode == -1) {
                    bestNode = cursor;
                    bestDist = dist;
                }

                // See if sibling subtree needs to be visited
                axis = KdTreeMemPool[cursor].splitPlane;
                dist = MOD(nodes[KdTreeMemPool[cursor].clusterId].pt.loc[axis]-point.loc[axis]);
                if (dist < bestDist) {
                    // Traverse down the the other subtree.
                    if (KdTreeMemPool[cursor].leftChildIndex == prevCursor && 
                        KdTreeMemPool[cursor].rightChildIndex != -1)
                    {
                        subRoot = KdTreeMemPool[cursor].rightChildIndex;
                        break;
                    } else if (KdTreeMemPool[cursor].leftChildIndex != -1) {
                        subRoot = KdTreeMemPool[cursor].leftChildIndex;
                        break;
                    }
                }
            }
            // cursor = parent(cursor)
            prevCursor = cursor;
            cursor = KdTreeMemPool[cursor].parentIndex;
        }
        if (cursor == -1)
            break;
        // Else, traverse down new subRoot and up again.
    }
    assert(bestNode != -1 && bestDist >= 0);
    return &nodes[KdTreeMemPool[bestNode].clusterId];
}
#endif // CPU_KD

#endif // KD_C
